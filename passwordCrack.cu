
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstring>
#include<ctime>
#include<cstdlib>
using namespace std;

int main()
{
int a[1000]; //array of all possible password characters
int length; //length of password
int random; //random password to be generated
int b[1000] = { 48 }; //array of attempted password cracks
unsigned long long tries = 0;
bool cracked = false;

cout << "Enter a password length: ";
cin >> length;

srand(time(NULL));
for (int i =0; i<length; i++){
    
        random = (rand()%75)+48; 
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << endl;

do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 75 + 48){ 
            b[i] -= 75; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

cout << "\r       \n";
for(int i=0; i<length; i++)
    cout << char (b[i]);
cout << "\nNumber of tries: " << tries << endl;

return 0;
}
