
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstring>
#include<ctime>
#include<cstdlib>
using namespace std;

int main()
{
int a[1000]; //array of all possible password characters
int length; //length of password
int random; //random password to be generated
int b[1000] = { 48 }; //array of attempted password cracks
unsigned long long tries = 0;
bool cracked = false;

cout << "Enter a password length: ";
cin >> length;

srand(time(NULL));
for (int i =0; i<length; i++){
    
        random = (rand()%75)+48; //whatever the random number is 
        //when mod 94(printable characters), the result is never more 
        //than adding 33 non-printing characters to get 
        //128 total characters on the ASCII table
        //rand() gives an integer from 0 to max (32767)
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << endl;

do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 75 + 48){ //if the index in b array is more than 127 characters
            b[i] -= 75; //then decrement it so that the index in b array can be less than 94
                  //printable characters
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

cout << "\r       \n";
for(int i=0; i<length; i++)
    cout << char (b[i]);
cout << "\nNumber of tries: " << tries << endl;

return 0;
}
