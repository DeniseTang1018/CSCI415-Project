
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstring>
#include<ctime>
#include<cstdlib>
using namespace std;

int a[1000]; //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
int length; //length of password
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };


void serial_passwordCrack(){
bool cracked = false;
do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 26 + alphabet[i]){ 
            b[i] -= 26; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

}


__global__ void parallel_passwordCrack()
{

}

int main()
{
int random; //random password to be generated

cout << "Enter a password length: ";
cin >> length;


srand(time(NULL));
for (int i =0; i<length; i++){
    
        random = alphabet[(rand()%26)]; 
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << endl;
serial_passwordCrack();
cout << "\n";
for(int i=0; i<length; i++)
    cout << char (b[i]);
cout << "\nNumber of tries: " << tries << endl;

return 0;
}
