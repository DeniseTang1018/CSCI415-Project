
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<cstring>
#include<ctime>
#include<cstdlib>
#include<sys/time.h>
#include<stdio.h>
#include<iomanip>

using namespace std;

int a[1000]; //array of all possible password characters
int b[1000]; //array of attempted password cracks
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };


void serial_passwordCrack(int length){
bool cracked = false;
do{
    b[0]++;
    for(int i =0; i<length; i++){
        if (b[i] >= 26 + alphabet[i]){ 
            b[i] -= 26; 
            b[i+1]++;
        }else break;
    }
    cracked=true;
    for(int k=0; k<length; k++)
        if(b[k]!=a[k]){
            cracked=false;
            break;
        }
    if( (tries & 0x7ffffff) == 0 )
        cout << "\r       \r   ";
    else if( (tries & 0x1ffffff) == 0 )
        cout << ".";
    tries++;
}while(cracked==false);

}


__global__ void parallel_passwordCrack(int length,int*d_output,int* a, long attempts )
{	
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	bool cracked = false;
	//int mark=0;
        char alphabetTable[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };        
	int newB[1000]; 
	__shared__ int nIter;
	do{
	attempts++;
	newB[0]++;
	for(int i =0; i<length; i++){
        if (newB[i] >= 26 + alphabetTable[i]){ 
            newB[i] -= 26; 
            newB[i+1]++;
        }else break;
    }
    
    cracked=true;
    for(int k=0; k<length; k++)
    {
        if(newB[k]!=a[k]){
            cracked=false;
            break;
        }else;
    }
	if(cracked){
      __syncthreads();
     //idT = idx;
      nIter = 1;
       __syncthreads();
      break;

    }
		
	}while(!cracked);
	
	for(int i = 0; i< length; i++){
  
             d_output[i] = newB[i];
    }
/*do{
    newB[0]++;
    	
	if(mark<length){
        if (newB[idx] >= 26 + alphabetTable[idx]){ 
            newB[idx] -= 26; 
            newB[idx+1]++;
	}
		mark++;
	}else{
		//mark++;
		//mark=0;
	}
	
    cracked=true;
    for(int k=0; k<length; k++)
        if(newB[k]!=a[k]){
            cracked=false;
            break;
        }else//{cracked = true;}
//    if( (tries & 0x7ffffff) == 0 )
//        cout << "\r       \r   ";
//    else if( (tries & 0x1ffffff) == 0 )
//        cout << ".";
    attempts++;
}while(cracked==false);

	d_output[idx] = newB[idx];*/
	

}


long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}



int main()
{
int length; //length of password
int random; //random password to be generated
long attempts =0; //number of attempts to crack the password
int *d_input;

cout << "Enter a password length: ";
cin >> length;
int *h_gpu_result = (int*)malloc(1000*sizeof(int));

srand(time(NULL));
cout << "Random generated password: " << endl;
for (int i =0; i<length; i++){
    
        random = alphabet[(rand()%26)]; 
    
    a[i] = random; //adding random password to array
    cout << char(a[i]);
}cout << "\n" << endl;

//declare GPU memory pointers
  int *d_output;
//allocate GPU memory
hipMalloc((void **) &d_output,1000*sizeof(int));
hipMalloc((void **) &d_input,1000*sizeof(int));
//transfer the array to the GP
hipMemcpy(d_input, &a, 1000*sizeof(int),hipMemcpyHostToDevice);
//launch the kernel
//int threards = length/1024;
//parallel_passwordCrack<<<threards,1024>>>(length,d_output,d_input,attempts);
parallel_passwordCrack<<<length,1024>>>(length,d_output,d_input,attempts);
//copy back the result array to the CPU
hipMemcpy(h_gpu_result,d_output,1000*sizeof(int),hipMemcpyDeviceToHost);

cout << "Serial Password Cracked: " << endl;
serial_passwordCrack(length);
cout << "\n";
for(int i=0; i<length; i++){
    cout << char(b[i]);
}cout << "\nNumber of tries: " << tries << endl;

cout << "\nParallel Password Cracked: " << endl;
for(int i=0; i<length; i++){
	cout << char(h_gpu_result[i]);
}
cout << "\nNumber of attempts: " << attempts << endl;

hipFree(d_output);
hipFree(d_input);
free(h_gpu_result);

return 0;
}
